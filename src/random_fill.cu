#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#include <omp.h>

#include "utils.hpp"

#define NUM_ITERATION   10

#ifndef DATA_T
#define DATA_T          float
#endif
#define STRINGIFY(s)    #s
#define MACRO_STR(m)   STRINGIFY(m)

int main(int argc, const char * argv[]) {
    DATA_T * X_h, * X_d;
    int N;

    bool is_data_type_valid = 
        std::is_same<DATA_T, float>::value || std::is_same<DATA_T, double>::value;

    if (argc > 1 && is_data_type_valid) {
        N = std::stoi(argv[1]);
    }
    else {
        perror("Usage: ./fma_vector <vector size>\n");
        exit(1);
    }

    hipDeviceProp_t prop = utils::get_device_properties();
    int num_threads_per_block = std::gcd(prop.maxThreadsPerBlock, prop.maxThreadsPerMultiProcessor);
    // why 4? Increasing the number of blocks per SM can hide latency. 
    // Larger number is also better for throughput, especially for longer inputs.
    int num_blocks_per_sm = (prop.maxThreadsPerMultiProcessor / num_threads_per_block) * 4;
    int num_blocks = prop.multiProcessorCount * num_blocks_per_sm;

    std::string bytes_s = utils::formatBytes(N * sizeof(DATA_T));

    printf("Filling Vector with Random Values\n");
    printf("- data type: %s\n", MACRO_STR(DATA_T));
    printf("- vector size: %d (%s)\n", N, bytes_s.c_str());
    printf("- #blocks: %d\n", num_blocks);
    printf("- #threads per block: %d\n", num_threads_per_block);

    if (num_blocks * num_threads_per_block > N) {
        perror("The vector size is too small for the current device configuration.\n");
        exit(1);
    }

    X_h = new DATA_T[N];

    hipMalloc((void **)&X_d, sizeof(DATA_T) * N);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    utils::random_fill_d<DATA_T>(X_d, N);
    hipMemcpyAsync(X_h, X_d, sizeof(DATA_T) * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    double total_time = 0.0;

    for (int i = 0; i < NUM_ITERATION; i++) {
        hipEventRecord(start);

        utils::random_fill_d<DATA_T>(X_d, N);
        // utils::random_fill_h<DATA_T>(X_h, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float t;
        hipEventElapsedTime(&t, start, stop); 

        total_time += t;
    }

    printf("Average time: %.6f (ms)\n", total_time / NUM_ITERATION);

    delete[] X_h;
    hipFree(X_d);

    return 0;
}